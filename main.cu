#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <vector>
#include "utils/logger.cuh"
#include "utils/cuda_utils.cuh"
#include "utils/timer.h"
#include "defs.cuh"
#include "LAP/device_utils.cuh"
#include "LAP/Hung_lap.cuh"
#include "LAP/lap_kernels.cuh"

#include "QAP/config.h"
#include "QAP/problem_generator.h"
#include "QAP/gurobi_solver.h"
// #include "RCAP/rcap_functions.cuh"

#include <queue>

int main(int argc, char **argv)
{
  Log(info, "Starting program");
  Config config = parseArgs(argc, argv);

  problem_info *h_problem_info = generate_problem(config, config.seed);
  print(h_problem_info, true, true);
  printConfig(config);

  if (config.user_n > 50)
  {
    Log(critical, "Problem size too large, Implementation not ready yet. Use problem size <= 50");
    exit(-1);
  }

#if 0

  Timer t = Timer();

  // Solve RCAP
  cost_type UB = solve_with_gurobi<cost_type, weight_type>(h_problem_info->costs, h_problem_info->weights, h_problem_info->budgets, psize, ncommodities);
  Log(info, "RCAP solved with GUROBI: objective %u\n", (uint)UB);

  Log(info, "Time taken by Gurobi: %f sec", t.elapsed());

  Log(debug, "Solving RCAP with Branching");
  t.reset();

  // Define a heap from the standard priority queue package
  std::priority_queue<node, std::vector<node>, std::greater<node>> heap;
  bnb_stats stats = bnb_stats();

  node_info *root_info = new node_info(psize);
  root_info->LB = 0;
  root_info->level = 0;
  node root = node(0, root_info);
  heap.push(root);
  stats.max_heap_size = max(stats.max_heap_size, (uint)heap.size());
  // start branch and bound
  bool optimal = false;
  node opt_node = node(0, new node_info(psize));
  // uint iter = 0;

  // Log(debug, " Subgrad on root node");
  // root.key = update_bounds_subgrad(h_problem_info, root, UB);
  // exit(-1);
  do
  {
    // Log(debug, "Starting iteration# %u", iter++);
    // get the best node from the heap
    node best_node = node(0, new node_info(psize));
    best_node.copy(heap.top(), psize);
    delete heap.top().value;
    heap.pop();
    // Log(info, "best node key %u", (uint)best_node.key);
    // bool feasible = feas_check_naive(h_problem_info, best_node);
    bool feasible = feas_check(h_problem_info, best_node);
    if (feasible)
    {
      // Update bound of the best node
      // update_bounds(h_problem_info, best_node);
      best_node.key = update_bounds_subgrad(h_problem_info, best_node, UB);

      uint level = best_node.value->level;
      if (best_node.key <= UB && best_node.value->level == psize)
      {
        optimal = true;
        Log(critical, "Optimality Reached");
        opt_node.copy(best_node, psize);
        delete best_node.value;
        break;
      }
      else if (best_node.key <= UB)
      {
        stats.nodes_explored++;
        // Branch on the best node to create (psize - level) new children nodes
        for (uint i = 0; i < psize - level; i++)
        {
          // Create a new child node
          node_info *child_info = new node_info(psize);
          child_info->LB = best_node.value->LB;
          child_info->level = level + 1;
          for (uint j = 0; j < psize; j++)
          {
            child_info->fixed_assignments[j] = best_node.value->fixed_assignments[j];
          }

          // Update fixed assignments of the child by updating the ith unassigned assignment to level
          uint counter = 0;
          for (uint index = 0; index < psize; index++)
          {
            if (counter == i && child_info->fixed_assignments[index] == -1)
            {
              // Log(debug, "Code reached here\n");
              child_info->fixed_assignments[index] = level;

              break;
            }
            if (child_info->fixed_assignments[index] == -1)
              counter++;
          }

          node child = node(best_node.key, child_info);
          heap.push(child);
        }
        stats.max_heap_size = max(stats.max_heap_size, (uint)heap.size());
      }
      else
      {
        // Prune the node
        stats.nodes_pruned_incumbent++;
      }
    }
    else
    {
      if (heap.size() <= 0)
      {
        Log(critical, "Heap underflow");
        exit(-1);
      }
      // Prune the node
      stats.nodes_pruned_infeasible++;
    }
    delete best_node.value;
  } while (!optimal || !heap.empty());

  if (optimal)
  {
    Log(critical, "Optimal solution found with objective %u", (uint)opt_node.key);
  }
  else
  {
    Log(critical, "Optimal solution not found");
  }
  Log(info, "Max heap size during execution: %lu", stats.max_heap_size);
  Log(info, "Nodes Explored: %u, Incumbant: %u, Infeasible: %u", stats.nodes_explored, stats.nodes_pruned_incumbent, stats.nodes_pruned_infeasible);

  Log(info, "Exiting program");
  Log(info, "Total time taken: %f sec", t.elapsed());

  delete h_problem_info;
  while (!heap.empty())
  {
    delete heap.top().value;
    heap.pop();
  }
  delete opt_node.value;
#endif
  return 0;
}